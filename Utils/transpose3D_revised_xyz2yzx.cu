
#include <hip/hip_runtime.h>
#ifndef MY_CUDA_BLOCK_DIM
#define MY_CUDA_BLOCK_DIM 16
#endif

__global__ void  transpose3D_revised_xyz2yzx( float *odata, float *idata, 
    unsigned int n1, unsigned int n2, unsigned int n3, 
    unsigned int Gx, unsigned int Gz, 
    float one_over_Gx, float one_over_Gz, unsigned int k2 )
{
    __shared__ float block[MY_CUDA_BLOCK_DIM][MY_CUDA_BLOCK_DIM+1];

    float tmp1 ;
    unsigned int s1, s2, t1, t2 ;
    unsigned int xIndex, yIndex, zIndex ;
    unsigned int index_in, index_out ;
	
    tmp1 = __uint2float_rz( blockIdx.x ) ;
    tmp1 = floorf( tmp1 * one_over_Gz ) ;
    s1 = __float2uint_rz( tmp1 ) ; 
    t1 = blockIdx.x - Gz*s1 ;
 	
    tmp1 = __uint2float_rz( blockIdx.y ) ;
    tmp1 = floorf( tmp1 * one_over_Gx ) ;
    s2 = __float2uint_rz( tmp1 ) ; 
    t2 = blockIdx.y - Gx*s2 ;
 
    yIndex = s2*k2 + s1 ;
 
    zIndex = t1 * MY_CUDA_BLOCK_DIM + threadIdx.x ;
    xIndex = t2 * MY_CUDA_BLOCK_DIM + threadIdx.y ;

    if ( (yIndex < n2) && (xIndex < n1) && (zIndex < n3)  ){
        index_in = (xIndex * n2 + yIndex) * n3 + zIndex ; 
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }
    __syncthreads();

    xIndex = t2 * MY_CUDA_BLOCK_DIM + threadIdx.x ;
    zIndex = t1 * MY_CUDA_BLOCK_DIM + threadIdx.y ;
 	
    if ( (yIndex < n2) && (xIndex < n1) && (zIndex < n3)  ){
        index_out = (yIndex * n3 + zIndex) * n1 + xIndex ; 
        odata[index_out] = block[threadIdx.x][threadIdx.y] ;
    } 	
 
}