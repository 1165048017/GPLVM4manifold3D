
#include <hip/hip_runtime.h>
__global__ void copyFFTtoDCT(float *out, float *in)
{
	unsigned int lineId, lineSize, valSrcId, valDstId;
	
	lineId = 2 * blockIdx.x;
	valSrcId = threadIdx.x;
	lineSize = blockDim.x;
	
	if (valSrcId % 2 == 0)
	{ valDstId = valSrcId - floor(valSrcId/2.0f); }
	else
	{ valDstId = lineSize - floor(valSrcId/2.0f) - 1; }
	
	//out[lineId * lineSize + valSrcId] = lineId * lineSize + valDstId;
	out[lineId * lineSize + valSrcId] = in[lineId * lineSize + valDstId];
	
	if (valSrcId % 2 == 0)
	{ valDstId = lineSize + valSrcId - floor(valSrcId/2.0f); }
	else
	{ valDstId = lineSize + lineSize - floor(valSrcId/2.0f) - 1; }
	
	//out[lineId * lineSize + valSrcId + lineSize] = lineId * lineSize + valDstId;
	out[lineId * lineSize + valSrcId + lineSize] = in[lineId * lineSize + valDstId];
}